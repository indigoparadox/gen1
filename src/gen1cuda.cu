#include "hip/hip_runtime.h"

#include "gen1.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>

__global__
void setup_curand( int* init, hiprandState *state ) {
   int idx = threadIdx.x;

   hiprand_init( *init, idx, 0, &state[idx] );
}

__global__
void generate_line_c( hiprandState* state, uint8_t* buffer, int width ) {
   int idx = threadIdx.x;
   int i = 0;
   float f_rand;

   for( i = 0 ; width > i ; i++ ) {
      f_rand = hiprand_uniform( state + idx );
      f_rand *= (UINT8_MAX + 0.999999);

      buffer[i] = (uint8_t)truncf( f_rand );
   }
}

extern "C" void generate_line( uint8_t* buffer, int width ) {
   hiprandState *d_state = NULL;
   int init_time = 0;

   hipMalloc( &d_state, sizeof( hiprandState ) * 256 );

   init_time = time( NULL );

   setup_curand<<<1, 1>>>( &init_time, d_state );

   generate_line_c<<<0, 256>>>( d_state, buffer, width );

   hipFree( d_state );
}

__global__
void fitness_score_add(
   float* out, const uint8_t* buffer_tgt,
   const uint8_t* buffer_test, int byte_width
) {
   int i = threadIdx.x;
   int stride = blockDim.x;

   for( ; byte_width > i ; i += stride ) {
      *out += abs( (buffer_tgt[i] - buffer_test[i]) );
   }
}

__global__
void combine_lines_c( uint8_t* line_dest, const uint8_t* line_src, int width ) {
   int i = threadIdx.x;
   int stride = blockDim.x;

   for( ; width > i ; i += stride ) {
      if( 0 == i % 2 ) {
         line_dest[i] = line_src[i];
      }
   }
}

extern "C" float fitness_score(
   int width, const uint8_t* buffer_tgt, const uint8_t* buffer_test
) {
   uint8_t* d_line_master = NULL;
   uint8_t* d_candidate = NULL;
   float* d_score_total = NULL;
   float out = 0;

   hipMalloc(
      (void**)&d_score_total, sizeof( float ) );
   hipMalloc(
      (void**)&d_candidate, sizeof( uint8_t ) * width );
   hipMalloc(
      (void**)&d_score_total, sizeof( uint8_t ) * width );

   hipMemcpy( d_candidate, buffer_test, width, hipMemcpyHostToDevice );
   hipMemcpy( d_line_master, buffer_tgt, width, hipMemcpyHostToDevice );

   fitness_score_add<<<0, 256>>>(
      d_score_total, d_line_master, d_candidate, width );

   hipMemcpy( &out, d_score_total, sizeof( float ), hipMemcpyDeviceToHost );

   hipFree( d_line_master );
   hipFree( d_candidate );
   hipFree( d_score_total );

   return out;
}

extern "C"
void combine_lines( uint8_t* line_dest, const uint8_t* line_src, int width ) {
   uint8_t* d_source = NULL;
   uint8_t* d_dest = NULL;

   hipMalloc(
      (void**)&d_source, sizeof( uint8_t ) * width );
   hipMalloc(
      (void**)&d_dest, sizeof( uint8_t ) * width );

   hipMemcpy( d_source, line_src, width, hipMemcpyHostToDevice );
   hipMemcpy( d_dest, line_dest, width, hipMemcpyHostToDevice );

   combine_lines_c<<<0, 256>>>( d_dest, d_source, width );

   hipMemcpy(
      &line_dest, d_dest, sizeof( uint8_t ) * width, hipMemcpyDeviceToHost );

   hipFree( d_source );
   hipFree( d_dest );
}

